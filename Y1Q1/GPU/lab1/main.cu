/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"

int main (int argc, char *argv[])
{
    Timer timer;
    hipError_t cuda_ret;
    hipError_t err;

    // Initialize host variables ----------------------------------------------
    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./sgemm                # All matrices are 1000 x 1000"
           "\n    Usage: ./sgemm <m>            # All matrices are m x m"
           "\n    Usage: ./sgemm <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
           "\n");
        exit(0);
    }

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    A_h = (float*) malloc( sizeof(float) * A_sz );
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float) * B_sz );
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float) * C_sz );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol, matBrow, matBcol, matArow, matBcol);

    // Allocate device variables ----------------------------------------------
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    // INSERT CODE HERE
    // hipMalloc calls to allocate memory in the device...
	err = hipMalloc((void**) &A_d, sizeof(float) * A_sz);
	if (err!=hipSuccess) {
		printf("\nChecking hipMalloc for A... %s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	} 
	err = hipMalloc((void**) &B_d, sizeof(float) * B_sz);
	if (err!=hipSuccess) {
		printf("\nChecking hipMalloc for B... %s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	} 
	err = hipMalloc((void**) &C_d, sizeof(float) * C_sz);
	if (err!=hipSuccess) {
		printf("\nChecking hipMalloc for C... %s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	} 
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    // INSERT CODE HERE
    // hipMemcpy calls to send variables to the device...
	err = hipMemcpy(A_d, A_h, sizeof(float) * A_sz, hipMemcpyHostToDevice);
	if (err!=hipSuccess) {
		printf("Checking hipMemcpy for A... %s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	} 
	err = hipMemcpy(B_d, B_h, sizeof(float) * B_sz, hipMemcpyHostToDevice);
	if (err!=hipSuccess) {
		printf("Checking hipMemcpy for B... %s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	} 
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard sgemm interface
    // Let's run the kernel...
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    basicSgemm('N', 'N', matArow, matBcol, matBrow, 1.0f, A_d, matArow, B_d, matBrow, 0.0f, C_d, matBrow);
    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables to host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    // INSERT CODE HERE
    // Sending back the results...
	err = hipMemcpy(C_h, C_d, sizeof(float) * C_sz, hipMemcpyDeviceToHost);
	if (err!=hipSuccess) {
		printf("Got it!!! %s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	} 
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------
    printf("Verifying results..."); fflush(stdout);
    // verify(A_h, B_h, C_h, matArow, matAcol, matBcol);
    
    // Printing relatively small matrices just for testing purposes...
    if(matArow * matBcol < 100){ 
		for(int i = 0; i < A_sz; ++i){
			if(i % matAcol == 0){
				printf("\n");
			} 
			printf("%.2f\t", A_h[i]);
		}
		printf("\n");
		for(int i = 0; i < B_sz; ++i){
			if(i % matBcol == 0){
				printf("\n");
			} 
			printf("%.2f\t", B_h[i]);
		}
		printf("\n");
		for(int i = 0; i < C_sz; ++i){
			if(i % matBcol == 0){
				printf("\n");
			} 
			printf("%.2f\t", C_h[i]);
		}
		printf("\n");
	}
	
    // Free memory ------------------------------------------------------------
    free(A_h);
    free(B_h);
    free(C_h);

    // INSERT CODE HERE
    // hipFree calls to free memory...
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

    return 0;
}

