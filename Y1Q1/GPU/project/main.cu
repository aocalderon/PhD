/******************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"

int main (int argc, char *argv[])
{
    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h, *D;
    float *A_d, *B_d, *C_d;
    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000"
      "\n    Usage: ./sgemm-tiled <m>            # All matrices are m x m"
      "\n    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
      "\n");
        exit(0);
    }

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float)*B_sz );
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*C_sz );
    
    D = (float*) malloc( sizeof(float)*C_sz );
    

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol, matBrow, matBcol, matArow, matBcol);

    // Allocate device variables

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    cuda_ret = hipMalloc((void **) &A_d, sizeof(float) * A_sz);
    if(cuda_ret != hipSuccess){
		printf("\nChecking hipMalloc for A... %s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
    }
    cuda_ret = hipMalloc((void **) &B_d, sizeof(float) * B_sz);
    if(cuda_ret != hipSuccess){
    	printf("\nChecking hipMalloc for B...  %s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
    }
    cuda_ret = hipMalloc((void **) &C_d, sizeof(float) * C_sz);
    if(cuda_ret != hipSuccess){
    	printf("\nChecking hipMalloc for C...  %s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    cuda_ret = hipMemcpy(A_d, A_h, sizeof(float) * A_sz, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess){
    	printf("\nChecking hipMemcpy for A...  %s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
    }
    cuda_ret = hipMemcpy(B_d, B_h, sizeof(float) * B_sz, hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess){
    	printf("\nChecking hipMemcpy for A...  %s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard sgemm interface

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    basicSgemm('N', 'N', matArow, matBcol, matBrow, 1.0f, A_d, matArow, B_d, matBrow, 0.0f, C_d, matBrow);

    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    cuda_ret = hipMemcpy(C_h, C_d, sizeof(float) * C_sz, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess){
    	printf("\nChecking hipMemcpy for C...  %s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, matArow, matAcol, matBcol, D);

    // Printing relatively small matrices just for testing purposes...
    if(matArow * matBcol <= 100){
		for(int i = 0; i < A_sz; ++i){
			if(i % matAcol == 0){
				printf("\n");
			}
			printf("%.2f\t", A_h[i]);
		}
		printf("\n");
		for(int i = 0; i < B_sz; ++i){
			if(i % matBcol == 0){
				printf("\n");
			}
			printf("%.2f\t", B_h[i]);
		}
		printf("\n");
		for(int i = 0; i < C_sz; ++i){
			if(i % matBcol == 0){
				printf("\n");
			}
			printf("%.2f\t", C_h[i]);
		}
		printf("\n");
		for(int i = 0; i < C_sz; ++i){
			if(i % matBcol == 0){
				printf("\n");
			}
			printf("%.2f\t", D[i]);
		}
		printf("\n");
	}

    // Free memory

    free(A_h);
    free(B_h);
    free(C_h);
    free(D);

    //INSERT CODE HERE
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return 0;
}
