
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
	
	__shared__ float partialSum[2 * BLOCK_SIZE];
	unsigned int t = threadIdx.x;
	unsigned int start = 2 * blockIdx.x * blockDim.x;
	// unsigned int c = blockDim.x * blockIdx.x + t;	

	if(start + t < size)
		partialSum[t] = in[start + t];
	else
		partialSum[t] = 0.0f;
	
	if(start + blockDim.x + t < size)
		partialSum[blockDim.x + t] = in[start + blockDim.x + t];
	else
		partialSum[blockDim.x + t] = 0.0f;

	for (int stride = blockDim.x; stride > 0; stride /= 2){
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t + stride];
	} 
	__syncthreads();

	out[blockIdx.x] = partialSum[0];
	
}
